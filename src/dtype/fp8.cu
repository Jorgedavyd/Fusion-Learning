#include "hip/hip_runtime.h"
/* Features of Fusion floats:
Always parse by copy on instance for broadcasting?.
Callable just on device.
Move not allowed.
Copy allowed.
Ensure coalescing patterns
*/

#include <cassert>
#include <cstdio>
#include "Fusion/errorMacros.h"
#include "floatBase.h"
#include <hip/hip_fp8.h>

template <typename T, unsigned int align_ = 1>
class SPBase : private FusionFloatBase<T, align_> {
public:
    __device__ __forceinline__ SPBase<T, align_> operator*(SPBase<T, align_>&& a) const {
        return SPBase<T, align_>(_data * a.get_value());
    }

    __device__ __forceinline__ SPBase<T, align_> operator+(SPBase<T, align_>&& a) const {
        return SPBase<T, align_>(_data + a.get_value());
    }

    __device__ __forceinline__ SPBase<T, align_> operator-(SPBase<T, align_>&& a) const {
        return SPBase<T, align_>(_data - a.get_value());
    }

    __device__ __forceinline__ SPBase<T, align_> operator/(SPBase<T, align_>&& a) const {
        return SPBase<T, align_>(_data / a.get_value());
    }
};

using fp8_e4m3 = SPBase<__hip_fp8_e4m3_fnuz, 1>;
using fp8_e5m2 = SPBase<__hip_fp8_e5m2_fnuz, 1>;
using fp8_x2_e4m3 = SPBase<__hip_fp8_e4m3_fnuz, 2>;
using fp8_x2_e5m2 = SPBase<__hip_fp8_e5m2_fnuz, 2>;
using fp8_x4_e4m3 = SPBase<__hip_fp8_e4m3_fnuz, 4>;
using fp8_x4_e5m2 = SPBase<__hip_fp8_e5m2_fnuz, 4>;
