#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include "fusion/tensor/DeviceDescriptor"
class FusionTensorDescriptor; // Creates optimal tensors from torch::Tensor
class FusionTensorAccessor; // Creates an accessor to the data through mappings from (n, c, w, h) to its optimal structure
// Asynchronous methods
template <typename scalar_t, size_t N, template <typename U> class PtrTraits = DefaultPtrTraits, typename index_t = int32_t>
// Being an interface between the optimal tensor and the kernels
class FusionTensorAccessor {
	protected:
		PtrTraits data_;
		size_t size_;
		size_t stride;

	public:
		const scalar_t& operator[](index_t ); // Define how to access data
};

// create coalesced data (one timer), just at the beginning
template <typename scalar_t, size_t N, template <typename U> class PtrTraits = DefaultPtrTraits, typename index_t = int32_t>
class FusionTensorDescriptor {
	protected:
		const torch::GenericPackedTensor<scalar_t, N, PtrTraits, index_t> accessor;
		// Given the input index_t and the new structure of the data it creates a mapping for the optimized data
		index_t idx_mapping (index_t idx) {
			return ...;
		} 
	public:
		FusionTensorBase(const torch::Tensor* tensor) {
			TORCH_CHECK(index_t == 32 || index_t == 64, "Not valid index type");
			CHECK(*tensor);
			// Create torch accessor to access the data to re allocate
			accessor = (index_t==32) ? tensor.packed_accessor32<scalar_t, N, PtrTraits> : tensor.packed_accessor64<scalar_t, N, PtrTraits>;
			// Optimize tensor allocation
			this->optimizeTensor();
			// Delete default tensor allocation (torch) (before sending from shared memory to global)	
		};
	private:
		//Purpose: Creating asynchronous data optimization step	
		__global__ void optimizeTensorKernelParent (void) {
		}
		
		__global__ void optimizeTensorKernelChild (void) {
		}
		
		static void optimizeTensor (void) {

		} // See if that would make optimizeTensor be called just once
		void to_host (void) {};
		void to_device (void) {};

}	
template <typename scalar_t, size_t N, template <typename U> class PtrTraits = DefaultPtrTraits, typename index_t = int32_t>
class FusionTensor: protected FusionTensorBase, protected FusionTensorAccessor {
	private:
	public:
		FusionTensorDescriptor (const torch::Tensor* tensor)  : FusionTensorBase(tensor), FusionTensorAccessor(tensor) {
		__device__ __forceinline__ T& operator[](index_t index) {
			return this->data_[this->strides_[0]*i];
		}

		}

};

template <>
using FusionTensorDescriptor32 = FusionTensorDescriptor<int32_t>;

template <>
using FusionTensorDescriptor64 = FusionTensorDescriptor<int64_t>;
